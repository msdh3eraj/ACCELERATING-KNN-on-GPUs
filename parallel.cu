
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define TRAIN_SIZE 100
#define TEST_SIZE 500
#define IMAGE_SIZE 64
#define K 1
#define CHUNK_SIZE (TEST_SIZE/32)

typedef struct {
    double *pixels;
    int label;
} Image;

__device__ double distanceBetweenImages(double *pixels1, double *pixels2) {
    double dist = 0;
    for (int i = 0; i < IMAGE_SIZE * IMAGE_SIZE; i++) {
        double diff = pixels1[i] - pixels2[i];
        dist += diff * diff;
    }
    return sqrt(dist);
}

__global__ void knn_slave(Image *train_data, Image *test_data, int *results) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int start = tid * CHUNK_SIZE;
    int end = start + CHUNK_SIZE;
    if (tid == 31) {
        end = TEST_SIZE;
    }
    for (int i = start; i < end; i++) {
        double distances[TRAIN_SIZE];
        int indices[TRAIN_SIZE];
        for (int j = 0; j < TRAIN_SIZE; j++) {
            distances[j] = distanceBetweenImages(train_data[j].pixels, test_data[i].pixels);
            indices[j] = j;
        }
        for (int j = 0; j < TRAIN_SIZE - 1; j++) {
            for (int k = j + 1; k < TRAIN_SIZE; k++) {
                if (distances[k] < distances[j]) {
                    double temp_dist = distances[j];
                    int temp_index = indices[j];
                    distances[j] = distances[k];
                    indices[j] = indices[k];
                    distances[k] = temp_dist;
                    indices[k] = temp_index;
                }
            }
        }

        int counts[2] = {0, 0};
        for (int j = 0; j < K; j++) {
            int idx = indices[j];
            counts[train_data[idx].label]++;
        }
        results[i] = (counts[0] > counts[1]) ? 0 : 1;
    }
}


void generate_images(Image images[], int num_images) {
   
    for (int i = 0; i < num_images; i++) {
        hipMallocManaged(&(images[i].pixels), IMAGE_SIZE * IMAGE_SIZE * sizeof(double));
        for (int j = 0; j < IMAGE_SIZE * IMAGE_SIZE; j++) {
            double pixel = (double) rand() / RAND_MAX;
            images[i].pixels[j] = pixel;
        }
        images[i].label = (rand() % 2);
    }
}

void free_images(Image images[], int num_images) {
    for (int i = 0; i < num_images; i++) {
        hipFree(images[i].pixels);
    }
}

int main() {
    Image *train_data, *test_data;
    int *results;
    hipMallocManaged(&train_data, TRAIN_SIZE * sizeof(Image));
    hipMallocManaged(&test_data, TEST_SIZE * sizeof(Image));
    hipMallocManaged(&results, TEST_SIZE * sizeof(int));
    generate_images(train_data, TRAIN_SIZE);
    generate_images(test_data, TEST_SIZE);

    knn_slave<<<1,32>>>(train_data, test_data, results);
    hipDeviceSynchronize();
    
    int num_correct = 0;
    for (int i = 0; i < TEST_SIZE; i++) {
        if (results[i] == test_data[i].label) {
            num_correct++;
        }
    }



    double accuracy = (double) num_correct / TEST_SIZE;
    printf("Accuracy: %f\n", accuracy);
    free_images(train_data, TRAIN_SIZE);
    free_images(test_data, TEST_SIZE);
    hipFree(train_data);
    hipFree(test_data);
    hipFree(results);
    return 0;
}